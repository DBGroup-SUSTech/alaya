#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT license.

#include "Basic/Console/console_V3.hpp"
#include "common_includes.h"
#include <boost/program_options.hpp>
#include <cstdint>
#include <cstdlib>

#include "index.h"
#include "disk_utils.h"
#include "math_utils.h"
#include "memory_mapper.h"
#include "partition.h"
#include "pq_flash_index.h"
#include "timer.h"
#include "percentile_stats.h"
#include "program_options_utils.hpp"
#include "utils.h"

#ifndef _WINDOWS
#include <sys/mman.h>
#include <sys/stat.h>
#include <unistd.h>
#include "linux_aligned_file_reader.h"

#include <boost/coroutine/all.hpp>
#else
#ifdef USE_BING_INFRA
#include "bing_aligned_file_reader.h"
#else
#include "windows_aligned_file_reader.h"
#endif
#endif

#include "MessageQueue/message_queue.hpp"
#include "main.hpp"

#define WARMUP false

namespace po = boost::program_options;

void print_stats(std::string category, std::vector<float> percentiles, std::vector<float> results)
{
    diskann::cout << std::setw(20) << category << ": " << std::flush;
    for (uint32_t s = 0; s < percentiles.size(); s++)
    {
        diskann::cout << std::setw(8) << percentiles[s] << "%";
    }
    diskann::cout << std::endl;
    diskann::cout << std::setw(22) << " " << std::flush;
    for (uint32_t s = 0; s < percentiles.size(); s++)
    {
        diskann::cout << std::setw(9) << results[s];
    }
    diskann::cout << std::endl;
}

template <typename T, typename LabelT = uint32_t>
int search_disk_index_mpi(diskann::Metric &metric, const std::string &index_path_prefix,
                          const std::string &result_output_prefix, uint32_t **&result_ids, float **&result_dists,
                          T *query, const size_t query_num, const size_t query_dim, const std::string gt_file,
                          const uint32_t num_threads, const uint32_t recall_at, const uint32_t beamwidth,
                          const uint32_t num_nodes_to_cache, const uint32_t search_io_limit,
                          const std::vector<uint32_t> &Lvec, const float fail_if_recall_below,
                          const std::vector<std::string> &query_filters, const bool use_reorder_data = false)
{
    uint32_t *gt_ids = nullptr;
    float *gt_dists = nullptr;
    size_t gt_num, gt_dim;
    size_t query_aligned_dim = ROUND_UP(query_dim, 8);

    bool filtered_search = false;


    bool calc_recall_flag = false;
    if (gt_file != std::string("null") && gt_file != std::string("NULL") && file_exists(gt_file))
    {
        diskann::load_truthset(gt_file, gt_ids, gt_dists, gt_num, gt_dim);
        if (gt_num != query_num)
        {
            diskann::cout << "Error. Mismatch in number of queries and ground truth data" << std::endl;
        }
        calc_recall_flag = true;
    }

    std::string warmup_query_file = index_path_prefix + "_sample_data.bin";
    std::shared_ptr<AlignedFileReader> reader = nullptr;
#ifdef _WINDOWS
#ifndef USE_BING_INFRA
    reader.reset(new WindowsAlignedFileReader());
#else
    reader.reset(new diskann::BingAlignedFileReader());
#endif
#else
    reader.reset(new LinuxAlignedFileReader());
#endif
    std::unique_ptr<diskann::PQFlashIndex<T, LabelT>> _pFlashIndex(
        new diskann::PQFlashIndex<T, LabelT>(reader, metric));

    int res = _pFlashIndex->load(num_threads, index_path_prefix.c_str());


    if (res != 0)
    {
        return res;
    }

    std::vector<uint32_t> node_list;
    diskann::cout << "Caching " << num_nodes_to_cache << " nodes around medoid(s)" << std::endl;
    _pFlashIndex->cache_bfs_levels(num_nodes_to_cache, node_list);
    // if (num_nodes_to_cache > 0)
    //     _pFlashIndex->generate_cache_list_from_sample_queries(warmup_query_file, 15, 6, num_nodes_to_cache,
    //     num_threads, node_list);
    _pFlashIndex->load_cache_list(node_list);
    node_list.clear();
    node_list.shrink_to_fit();


    omp_set_num_threads(num_threads);

    uint64_t warmup_L = 20;
    uint64_t warmup_num = 0, warmup_dim = 0, warmup_aligned_dim = 0;
    T *warmup = nullptr;

    if (WARMUP)
    {
        if (file_exists(warmup_query_file))
        {
            diskann::load_aligned_bin<T>(warmup_query_file, warmup, warmup_num, warmup_dim, warmup_aligned_dim);
        }
        else
        {
            warmup_num = (std::min)((uint32_t)150000, (uint32_t)15000 * num_threads);
            warmup_dim = query_dim;
            warmup_aligned_dim = query_aligned_dim;
            diskann::alloc_aligned(((void **)&warmup), warmup_num * warmup_aligned_dim * sizeof(T), 8 * sizeof(T));
            std::memset(warmup, 0, warmup_num * warmup_aligned_dim * sizeof(T));
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_int_distribution<> dis(-128, 127);
            for (uint32_t i = 0; i < warmup_num; i++)
            {
                for (uint32_t d = 0; d < warmup_dim; d++)
                {
                    warmup[i * warmup_aligned_dim + d] = (T)dis(gen);
                }
            }
        }
        diskann::cout << "Warming up index... " << std::flush;
        std::vector<uint64_t> warmup_result_ids_64(warmup_num, 0);
        std::vector<float> warmup_result_dists(warmup_num, 0);

#pragma omp parallel for schedule(dynamic, 1)
        for (int64_t i = 0; i < (int64_t)warmup_num; i++)
        {
            _pFlashIndex->cached_beam_search(warmup + (i * warmup_aligned_dim), 1, warmup_L,
                                             warmup_result_ids_64.data() + (i * 1),
                                             warmup_result_dists.data() + (i * 1), 4);
        }
        diskann::cout << "..done" << std::endl;
    }
    //=
    diskann::cout.setf(std::ios_base::fixed, std::ios_base::floatfield);
    diskann::cout.precision(2);

    std::string recall_string = "Recall@" + std::to_string(recall_at);
    diskann::cout << std::setw(6) << "L" << std::setw(12) << "Beamwidth" << std::setw(16) << "QPS" << std::setw(16)
                  << "Mean Latency" << std::setw(16) << "99.9 Latency" << std::setw(16) << "Mean IOs" << std::setw(16)
                  << "CPU (s)";
    if (calc_recall_flag)
    {
        diskann::cout << std::setw(16) << recall_string << std::endl;
    }
    else
        diskann::cout << std::endl;
    diskann::cout << "==============================================================="
                     "======================================================="
                  << std::endl;
    //=
    std::vector<std::vector<uint32_t>> query_result_ids(Lvec.size());
    std::vector<std::vector<float>> query_result_dists(Lvec.size());

    uint32_t optimized_beamwidth = 2;

    double best_recall = 0.0;

    result_ids = new uint32_t *[Lvec.size()];
    result_dists = new float *[Lvec.size()];

    for (uint32_t test_id = 0; test_id < Lvec.size(); test_id++)
    {
        uint32_t L = Lvec[test_id];

        if (L < recall_at)
        {
            diskann::cout << "Ignoring search with L:" << L << " since it's smaller than K:" << recall_at << std::endl;
            continue;
        }

        if (beamwidth <= 0)
        {
            diskann::cout << "Tuning beamwidth.." << std::endl;
            optimized_beamwidth =
                optimize_beamwidth(_pFlashIndex, warmup, warmup_num, warmup_aligned_dim, L, optimized_beamwidth);
        }
        else
            optimized_beamwidth = beamwidth;

        query_result_ids[test_id].resize(recall_at * query_num);
        query_result_dists[test_id].resize(recall_at * query_num);

        auto stats = new diskann::QueryStats[query_num];

        std::vector<uint64_t> query_result_ids_64(recall_at * query_num);
        auto s = std::chrono::high_resolution_clock::now();

#pragma omp parallel for schedule(dynamic, 1)
        for (int64_t i = 0; i < (int64_t)query_num; i++)
        {
            if (!filtered_search)
            {
                // query这个地方取的是一段地址内的数据。也就是一个query，对齐过后的query
                // query_result_ids_64.data() 是这个数组的首地址，而且这个数据是 uint32_t 类型
                _pFlashIndex->cached_beam_search(query + (i * query_aligned_dim), recall_at, L,
                                                 query_result_ids_64.data() + (i * recall_at),
                                                 query_result_dists[test_id].data() + (i * recall_at),
                                                 optimized_beamwidth, use_reorder_data, stats + i);
            }
            else
            {
                LabelT label_for_search;
                if (query_filters.size() == 1)
                { // one label for all queries
                    label_for_search = _pFlashIndex->get_converted_label(query_filters[0]);
                }
                else
                { // one label for each query
                    label_for_search = _pFlashIndex->get_converted_label(query_filters[i]);
                }
                _pFlashIndex->cached_beam_search(
                    query + (i * query_aligned_dim), recall_at, L, query_result_ids_64.data() + (i * recall_at),
                    query_result_dists[test_id].data() + (i * recall_at), optimized_beamwidth, true, label_for_search,
                    use_reorder_data, stats + i);
            }
        }
        auto e = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> diff = e - s;
        double qps = (1.0 * query_num) / (1.0 * diff.count());

        // query_result_ids_64 是本轮L产生的结果，然后将结果转移到query_result_ids[test_id]中去
        diskann::convert_types<uint64_t, uint32_t>(query_result_ids_64.data(), query_result_ids[test_id].data(),
                                                   query_num, recall_at);

        //=
        auto mean_latency = diskann::get_mean_stats<float>(
            stats, query_num, [](const diskann::QueryStats &stats) { return stats.total_us; });

        auto latency_999 = diskann::get_percentile_stats<float>(
            stats, query_num, 0.999, [](const diskann::QueryStats &stats) { return stats.total_us; });

        auto mean_ios = diskann::get_mean_stats<uint32_t>(stats, query_num,
                                                          [](const diskann::QueryStats &stats) { return stats.n_ios; });

        auto mean_cpuus = diskann::get_mean_stats<float>(stats, query_num,
                                                         [](const diskann::QueryStats &stats) { return stats.cpu_us; });
        if (test_id == Lvec.size() - 1)
            for (int ids_index = 0; ids_index < recall_at; ++ids_index)
                printf("before sending, in function.. result_ids in Lvec %d, is: %d\n", test_id,
                         query_result_ids[test_id][ids_index]);
        std::cout << std::endl;

        //= ==============================================计算recall============================================= =//
        // double recall = 0;
        // if (calc_recall_flag)
        // {
        //     //* 计算recall函数，返回值为recall，对每个L都计算一次。
        //     //* 传入参数为：query_num 查询个数、gt_ids, gt_dists, gt_dim,
        //     //* query_result_ids[test_id]这个是一维数组首地址，recall_at 就是K，所以参数传递没有问题。
        //     recall = diskann::calculate_recall((uint32_t)query_num, gt_ids, gt_dists, (uint32_t)gt_dim,
        //                                        query_result_ids[test_id].data(), recall_at, recall_at);
        //     // 在L个结果中，选出最佳recall
        //     best_recall = std::max(recall, best_recall);
        //     if constexpr (DEBUG)
        //     {
        //         Msg_warn("recall and best_recall is: %f, %f", recall, best_recall);
        //         for (int j = 0; j < query_result_ids[test_id].size(); ++j)
        //         {
        //             Msg_major("query_result_ids[%d][%d] = %d", test_id, j, query_result_ids[test_id][j]);
        //         }
        //         // Msg_major("query_result_ids[%zu][]")
        //     }
        // }

        // = 修改返回结果指针的指向

        result_ids[test_id] = new uint32_t[recall_at * query_num];
        memcpy(result_ids[test_id], query_result_ids[test_id].data(), recall_at * query_num * sizeof(uint32_t));

        result_dists[test_id] = new float[recall_at * query_num];
        memcpy(result_dists[test_id], query_result_dists[test_id].data(), recall_at * query_num * sizeof(float));

        diskann::cout << std::setw(6) << L << std::setw(12) << optimized_beamwidth << std::setw(16) << qps
                      << std::setw(16) << mean_latency << std::setw(16) << latency_999 << std::setw(16) << mean_ios
                      << std::setw(16) << mean_cpuus;
        if (calc_recall_flag)
        {
            diskann::cout << std::setw(16) << 0 << std::endl;
        }
        else
            diskann::cout << std::endl;
        delete[] stats;
    } // end of for Lvec.size()

    //= ==============================================存储结果到文件============================================= =//
    diskann::cout << "Done searching. Now saving results " << std::endl;
    uint64_t test_id = 0;
    for (auto L : Lvec)
    {
        if (L < recall_at)
            continue;

        std::string cur_result_path = result_output_prefix + "_" + std::to_string(L) + "_idx_uint32.bin";
        diskann::save_bin<uint32_t>(cur_result_path, query_result_ids[test_id].data(), query_num, recall_at);

        cur_result_path = result_output_prefix + "_" + std::to_string(L) + "_dists_float.bin";
        diskann::save_bin<float>(cur_result_path, query_result_dists[test_id++].data(), query_num, recall_at);
    }

    diskann::aligned_free(query);
    if (warmup != nullptr)
        diskann::aligned_free(warmup);

    // if constexpr (DEBUG)
    // {
    //     Msg_warn("result_num and result_dim is: %zu, %zu...", Lvec.size(), recall_at * query_num);
    //     for (int test_id = 0; test_id < Lvec.size(); test_id++)
    //     {
    //         for (int j = 0; j < recall_at * query_num; ++j)
    //         {
    //             Msg_major("after pointer reverted: result_ids[%d][%d] = %d", test_id, j, result_ids[test_id][j]);
    //         }
    //     }
    // }
    return best_recall >= fail_if_recall_below ? 0 : -1;
}

template <typename T, typename LabelT = uint32_t>
int search_disk_index(diskann::Metric &metric, const std::string &index_path_prefix,
                      const std::string &result_output_prefix, const std::string &query_file, std::string &gt_file,
                      const uint32_t num_threads, const uint32_t recall_at, const uint32_t beamwidth,
                      const uint32_t num_nodes_to_cache, const uint32_t search_io_limit,
                      const std::vector<uint32_t> &Lvec, const float fail_if_recall_below,
                      const std::vector<std::string> &query_filters, const bool use_reorder_data = false)
{
    diskann::cout << "Search parameters: #threads: " << num_threads << ", ";
    if (beamwidth <= 0)
        diskann::cout << "beamwidth to be optimized for each L value" << std::flush;
    else
        diskann::cout << " beamwidth: " << beamwidth << std::flush;
    if (search_io_limit == std::numeric_limits<uint32_t>::max())
        diskann::cout << "." << std::endl;
    else
        diskann::cout << ", io_limit: " << search_io_limit << "." << std::endl;

    std::string warmup_query_file = index_path_prefix + "_sample_data.bin";

    // load query bin
    T *query = nullptr;
    uint32_t *gt_ids = nullptr;
    float *gt_dists = nullptr;
    size_t query_num, query_dim, query_aligned_dim, gt_num, gt_dim;
    diskann::load_aligned_bin<T>(query_file, query, query_num, query_dim, query_aligned_dim);

    bool filtered_search = false;
    if (!query_filters.empty())
    {
        filtered_search = true;
        if (query_filters.size() != 1 && query_filters.size() != query_num)
        {
            std::cout << "Error. Mismatch in number of queries and size of query "
                         "filters file"
                      << std::endl;
            return -1; // To return -1 or some other error handling?
        }
    }

    bool calc_recall_flag = false;
    if (gt_file != std::string("null") && gt_file != std::string("NULL") && file_exists(gt_file))
    {
        diskann::load_truthset(gt_file, gt_ids, gt_dists, gt_num, gt_dim);
        if (gt_num != query_num)
        {
            diskann::cout << "Error. Mismatch in number of queries and ground truth data" << std::endl;
        }
        calc_recall_flag = true;
    }

    std::shared_ptr<AlignedFileReader> reader = nullptr;
#ifdef _WINDOWS
#ifndef USE_BING_INFRA
    reader.reset(new WindowsAlignedFileReader());
#else
    reader.reset(new diskann::BingAlignedFileReader());
#endif
#else
    reader.reset(new LinuxAlignedFileReader());
#endif

    std::unique_ptr<diskann::PQFlashIndex<T, LabelT>> _pFlashIndex(
        new diskann::PQFlashIndex<T, LabelT>(reader, metric));

    int res = _pFlashIndex->load(num_threads, index_path_prefix.c_str());

    if (res != 0)
    {
        return res;
    }

    std::vector<uint32_t> node_list;
    diskann::cout << "Caching " << num_nodes_to_cache << " nodes around medoid(s)" << std::endl;
    _pFlashIndex->cache_bfs_levels(num_nodes_to_cache, node_list);
    // if (num_nodes_to_cache > 0)
    //     _pFlashIndex->generate_cache_list_from_sample_queries(warmup_query_file, 15, 6, num_nodes_to_cache,
    //     num_threads, node_list);
    _pFlashIndex->load_cache_list(node_list);
    node_list.clear();
    node_list.shrink_to_fit();

    omp_set_num_threads(num_threads);

    uint64_t warmup_L = 20;
    uint64_t warmup_num = 0, warmup_dim = 0, warmup_aligned_dim = 0;
    T *warmup = nullptr;

    if (WARMUP)
    {
        if (file_exists(warmup_query_file))
        {
            diskann::load_aligned_bin<T>(warmup_query_file, warmup, warmup_num, warmup_dim, warmup_aligned_dim);
        }
        else
        {
            warmup_num = (std::min)((uint32_t)150000, (uint32_t)15000 * num_threads);
            warmup_dim = query_dim;
            warmup_aligned_dim = query_aligned_dim;
            diskann::alloc_aligned(((void **)&warmup), warmup_num * warmup_aligned_dim * sizeof(T), 8 * sizeof(T));
            std::memset(warmup, 0, warmup_num * warmup_aligned_dim * sizeof(T));
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_int_distribution<> dis(-128, 127);
            for (uint32_t i = 0; i < warmup_num; i++)
            {
                for (uint32_t d = 0; d < warmup_dim; d++)
                {
                    warmup[i * warmup_aligned_dim + d] = (T)dis(gen);
                }
            }
        }
        diskann::cout << "Warming up index... " << std::flush;
        std::vector<uint64_t> warmup_result_ids_64(warmup_num, 0);
        std::vector<float> warmup_result_dists(warmup_num, 0);

#pragma omp parallel for schedule(dynamic, 1)
        for (int64_t i = 0; i < (int64_t)warmup_num; i++)
        {
            _pFlashIndex->cached_beam_search(warmup + (i * warmup_aligned_dim), 1, warmup_L,
                                             warmup_result_ids_64.data() + (i * 1),
                                             warmup_result_dists.data() + (i * 1), 4);
        }
        diskann::cout << "..done" << std::endl;
    }

    diskann::cout.setf(std::ios_base::fixed, std::ios_base::floatfield);
    diskann::cout.precision(2);

    std::string recall_string = "Recall@" + std::to_string(recall_at);
    diskann::cout << std::setw(6) << "L" << std::setw(12) << "Beamwidth" << std::setw(16) << "QPS" << std::setw(16)
                  << "Mean Latency" << std::setw(16) << "99.9 Latency" << std::setw(16) << "Mean IOs" << std::setw(16)
                  << "CPU (s)";
    if (calc_recall_flag)
    {
        diskann::cout << std::setw(16) << recall_string << std::endl;
    }
    else
        diskann::cout << std::endl;
    diskann::cout << "==============================================================="
                     "======================================================="
                  << std::endl;

    std::vector<std::vector<uint32_t>> query_result_ids(Lvec.size());
    std::vector<std::vector<float>> query_result_dists(Lvec.size());

    uint32_t optimized_beamwidth = 2;

    double best_recall = 0.0;

    for (uint32_t test_id = 0; test_id < Lvec.size(); test_id++)
    {
        uint32_t L = Lvec[test_id];

        if (L < recall_at)
        {
            diskann::cout << "Ignoring search with L:" << L << " since it's smaller than K:" << recall_at << std::endl;
            continue;
        }

        if (beamwidth <= 0)
        {
            diskann::cout << "Tuning beamwidth.." << std::endl;
            optimized_beamwidth =
                optimize_beamwidth(_pFlashIndex, warmup, warmup_num, warmup_aligned_dim, L, optimized_beamwidth);
        }
        else
            optimized_beamwidth = beamwidth;

        query_result_ids[test_id].resize(recall_at * query_num);
        query_result_dists[test_id].resize(recall_at * query_num);

        auto stats = new diskann::QueryStats[query_num];

        std::vector<uint64_t> query_result_ids_64(recall_at * query_num);
        auto s = std::chrono::high_resolution_clock::now();

#pragma omp parallel for schedule(dynamic, 1)
        for (int64_t i = 0; i < (int64_t)query_num; i++)
        {
            if (!filtered_search)
            {
                // query这个地方取的是一段地址内的数据。也就是一个query，对齐过后的query
                // query_result_ids_64.data() 是这个数组的首地址，而且这个数据是 uint32_t 类型
                _pFlashIndex->cached_beam_search(query + (i * query_aligned_dim), recall_at, L,
                                                 query_result_ids_64.data() + (i * recall_at),
                                                 query_result_dists[test_id].data() + (i * recall_at),
                                                 optimized_beamwidth, use_reorder_data, stats + i);
            }
            else
            {
                LabelT label_for_search;
                if (query_filters.size() == 1)
                { // one label for all queries
                    label_for_search = _pFlashIndex->get_converted_label(query_filters[0]);
                }
                else
                { // one label for each query
                    label_for_search = _pFlashIndex->get_converted_label(query_filters[i]);
                }
                _pFlashIndex->cached_beam_search(
                    query + (i * query_aligned_dim), recall_at, L, query_result_ids_64.data() + (i * recall_at),
                    query_result_dists[test_id].data() + (i * recall_at), optimized_beamwidth, true, label_for_search,
                    use_reorder_data, stats + i);
            }
        }
        auto e = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> diff = e - s;
        double qps = (1.0 * query_num) / (1.0 * diff.count());

        diskann::convert_types<uint64_t, uint32_t>(query_result_ids_64.data(), query_result_ids[test_id].data(),
                                                   query_num, recall_at);

        auto mean_latency = diskann::get_mean_stats<float>(
            stats, query_num, [](const diskann::QueryStats &stats) { return stats.total_us; });

        auto latency_999 = diskann::get_percentile_stats<float>(
            stats, query_num, 0.999, [](const diskann::QueryStats &stats) { return stats.total_us; });

        auto mean_ios = diskann::get_mean_stats<uint32_t>(stats, query_num,
                                                          [](const diskann::QueryStats &stats) { return stats.n_ios; });

        auto mean_cpuus = diskann::get_mean_stats<float>(stats, query_num,
                                                         [](const diskann::QueryStats &stats) { return stats.cpu_us; });

        double recall = 0;
        if (calc_recall_flag)
        {
            recall = diskann::calculate_recall((uint32_t)query_num, gt_ids, gt_dists, (uint32_t)gt_dim,
                                               query_result_ids[test_id].data(), recall_at, recall_at);
            best_recall = std::max(recall, best_recall);
        }

        diskann::cout << std::setw(6) << L << std::setw(12) << optimized_beamwidth << std::setw(16) << qps
                      << std::setw(16) << mean_latency << std::setw(16) << latency_999 << std::setw(16) << mean_ios
                      << std::setw(16) << mean_cpuus;
        if (calc_recall_flag)
        {
            diskann::cout << std::setw(16) << recall << std::endl;
        }
        else
            diskann::cout << std::endl;
        delete[] stats;
    } // end of for Lvec.size()

    diskann::cout << "Done searching. Now saving results " << std::endl;
    // uint64_t test_id = 0;
    // for (auto L : Lvec)
    // {
    //     if (L < recall_at)
    //         continue;

    //     std::string cur_result_path = result_output_prefix + "_" + std::to_string(L) + "_idx_uint32.bin";
    //     diskann::save_bin<uint32_t>(cur_result_path, query_result_ids[test_id].data(), query_num, recall_at);

    //     cur_result_path = result_output_prefix + "_" + std::to_string(L) + "_dists_float.bin";
    //     diskann::save_bin<float>(cur_result_path, query_result_dists[test_id++].data(), query_num, recall_at);
    // }

    diskann::aligned_free(query);
    if (warmup != nullptr)
        diskann::aligned_free(warmup);
    return best_recall >= fail_if_recall_below ? 0 : -1;
}

int main(int argc, char **argv)
{
    // = 初始化MPI
    Env::initEnv(argc, argv);

    std::string data_type, dist_fn, index_path_prefix, result_path_prefix, query_file, gt_file, filter_label,
        label_type, query_filters_file;
    uint32_t num_threads, K, W, num_nodes_to_cache, search_io_limit;
    std::vector<uint32_t> Lvec;
    bool use_reorder_data = false;
    float fail_if_recall_below = 0.0f;

    po::options_description desc{
        program_options_utils::make_program_description("search_disk_index", "Searches on-disk DiskANN indexes")};
    try
    {
        desc.add_options()("help,h", "Print information on arguments");

        // Required parameters
        po::options_description required_configs("Required");
        required_configs.add_options()("data_type", po::value<std::string>(&data_type)->required(),
                                       program_options_utils::DATA_TYPE_DESCRIPTION);
        required_configs.add_options()("dist_fn", po::value<std::string>(&dist_fn)->required(),
                                       program_options_utils::DISTANCE_FUNCTION_DESCRIPTION);
        required_configs.add_options()("index_path_prefix", po::value<std::string>(&index_path_prefix)->required(),
                                       program_options_utils::INDEX_PATH_PREFIX_DESCRIPTION);
        required_configs.add_options()("result_path", po::value<std::string>(&result_path_prefix)->required(),
                                       program_options_utils::RESULT_PATH_DESCRIPTION);
        required_configs.add_options()("query_file", po::value<std::string>(&query_file)->required(),
                                       program_options_utils::QUERY_FILE_DESCRIPTION);
        required_configs.add_options()("recall_at,K", po::value<uint32_t>(&K)->required(),
                                       program_options_utils::NUMBER_OF_RESULTS_DESCRIPTION);
        required_configs.add_options()("search_list,L",
                                       po::value<std::vector<uint32_t>>(&Lvec)->multitoken()->required(),
                                       program_options_utils::SEARCH_LIST_DESCRIPTION);

        // Optional parameters
        po::options_description optional_configs("Optional");
        optional_configs.add_options()("gt_file", po::value<std::string>(&gt_file)->default_value(std::string("null")),
                                       program_options_utils::GROUND_TRUTH_FILE_DESCRIPTION);
        optional_configs.add_options()("beamwidth,W", po::value<uint32_t>(&W)->default_value(2),
                                       program_options_utils::BEAMWIDTH);
        optional_configs.add_options()("num_nodes_to_cache", po::value<uint32_t>(&num_nodes_to_cache)->default_value(0),
                                       program_options_utils::NUMBER_OF_NODES_TO_CACHE);
        optional_configs.add_options()(
            "search_io_limit",
            po::value<uint32_t>(&search_io_limit)->default_value(std::numeric_limits<uint32_t>::max()),
            "Max #IOs for search.  Default value: uint32::max()");
        optional_configs.add_options()("num_threads,T",
                                       po::value<uint32_t>(&num_threads)->default_value(omp_get_num_procs()),
                                       program_options_utils::NUMBER_THREADS_DESCRIPTION);
        optional_configs.add_options()("use_reorder_data", po::bool_switch()->default_value(false),
                                       "Include full precision data in the index. Use only in "
                                       "conjuction with compressed data on SSD.  Default value: false");
        optional_configs.add_options()("filter_label",
                                       po::value<std::string>(&filter_label)->default_value(std::string("")),
                                       program_options_utils::FILTER_LABEL_DESCRIPTION);
        optional_configs.add_options()("query_filters_file",
                                       po::value<std::string>(&query_filters_file)->default_value(std::string("")),
                                       program_options_utils::FILTERS_FILE_DESCRIPTION);
        optional_configs.add_options()("label_type", po::value<std::string>(&label_type)->default_value("uint"),
                                       program_options_utils::LABEL_TYPE_DESCRIPTION);
        optional_configs.add_options()("fail_if_recall_below",
                                       po::value<float>(&fail_if_recall_below)->default_value(0.0f),
                                       program_options_utils::FAIL_IF_RECALL_BELOW);

        // Merge required and optional parameters
        desc.add(required_configs).add(optional_configs);

        po::variables_map vm;
        po::store(po::parse_command_line(argc, argv, desc), vm);
        if (vm.count("help"))
        {
            std::cout << desc;
            return 0;
        }
        po::notify(vm);
        if (vm["use_reorder_data"].as<bool>())
            use_reorder_data = true;
    }
    catch (const std::exception &ex)
    {
        std::cerr << ex.what() << '\n';
        return -1;
    }

    diskann::Metric metric;
    if (dist_fn == std::string("mips"))
    {
        metric = diskann::Metric::INNER_PRODUCT;
    }
    else if (dist_fn == std::string("l2"))
    {
        metric = diskann::Metric::L2;
    }
    else if (dist_fn == std::string("cosine"))
    {
        metric = diskann::Metric::COSINE;
    }
    else
    {
        std::cout << "Unsupported distance function. Currently only L2/ Inner "
                     "Product/Cosine are supported."
                  << std::endl;
        return -1;
    }

    if ((data_type != std::string("float")) && (metric == diskann::Metric::INNER_PRODUCT))
    {
        std::cout << "Currently support only floating point data for Inner Product." << std::endl;
        return -1;
    }

    if (use_reorder_data && data_type != std::string("float"))
    {
        std::cout << "Error: Reorder data for reordering currently only "
                     "supported for float data type."
                  << std::endl;
        return -1;
    }

    if (filter_label != "" && query_filters_file != "")
    {
        std::cerr << "Only one of filter_label and query_filters_file should be provided" << std::endl;
        return -1;
    }

    std::vector<std::string> query_filters;
    if (filter_label != "")
    {
        query_filters.push_back(filter_label);
    }
    else if (query_filters_file != "")
    {
        query_filters = read_file_to_vector_of_strings(query_filters_file);
    }

    // = server0加载query file，gt file

    //*======================================================================================================================*/
    //*
    //*                                                     server0
    //*
    //*======================================================================================================================*/
    if (serverId() == 0)
    {
        float *query = nullptr;

        size_t query_num, query_dim, query_aligned_dim;
        uint32_t *gt_ids = nullptr;
        float *gt_dists = nullptr;
        size_t gt_num, gt_dim;
        diskann::load_aligned_bin<float>(query_file, query, query_num, query_dim, query_aligned_dim);
        // query_num = 1;
        uint32_t query_buffer_capacity = (query_num * query_dim * sizeof(data_type) + 2 * sizeof(size_t)) * 2;
        uint32_t result_buffer_capacity = (K * Lvec.size() * query_num * sizeof(data_type) + 2 * sizeof(size_t)) * 1.2;

        uint32_t beamwidth = W;
        uint32_t optimized_beamwidth = 2;

        // filter一直是空的，可以暂时先不管。
        bool filtered_search = false;
        if (!query_filters.empty())
        {
            filtered_search = true;
            if (query_filters.size() != 1 && query_filters.size() != query_num)
            {
                std::cout << "Error. Mismatch in number of queries and size of query "
                             "filters file"
                          << std::endl;
                return -1; // To return -1 or some other error handling?
            }
        }

        bool calc_recall_flag = false;
        if (gt_file != std::string("null") && gt_file != std::string("NULL") && file_exists(gt_file))
        {
            diskann::load_truthset(gt_file, gt_ids, gt_dists, gt_num, gt_dim);
            if (gt_num != query_num)
            {
                diskann::cout << "Error. Mismatch in number of queries and ground truth data" << std::endl;
            }
            calc_recall_flag = true;
        }
        auto s = std::chrono::high_resolution_clock::now();
        Message::MessageQueue<char> *msgq0 =
            new Message::MessageQueue<char>(query_buffer_capacity, result_buffer_capacity);
        msgq0->sendMessage<float>(1, query, query_num, query_dim, QUERY);
        msgq0->sendMessage<float>(2, query, query_num, query_dim, QUERY);
        msgq0->sendMessage<float>(3, query, query_num, query_dim, QUERY);
        auto e = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> send_time = e - s;

        uint32_t **result_ids2 = nullptr;
        float **result_dists2 = nullptr;
        size_t result_num2, result_dim2 = 0;
        auto before_recv3 = std::chrono::high_resolution_clock::now();

        msgq0->recvMessage<uint32_t>(3, result_ids2, result_num2, result_dim2, IDX);
        msgq0->recvMessage<float>(3, result_dists2, result_num2, result_dim2, DIST);

        auto after_recv3 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> recv3_time = after_recv3 - before_recv3;

        uint32_t **result_ids1 = nullptr;
        float **result_dists1 = nullptr;
        size_t result_num1, result_dim1 = 0;

        msgq0->recvMessage<uint32_t>(2, result_ids1, result_num1, result_dim1, IDX);
        msgq0->recvMessage<float>(2, result_dists1, result_num1, result_dim1, DIST);
        auto after_recv2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> recv2_time = after_recv2 - after_recv3;

        uint32_t **result_ids0 = nullptr;
        float **result_dists0 = nullptr;
        size_t result_num0, result_dim0 = 0;

        msgq0->recvMessage<uint32_t>(1, result_ids0, result_num0, result_dim0, IDX);
        msgq0->recvMessage<float>(1, result_dists0, result_num0, result_dim0, DIST);

        // if constexpr (DEBUG)
        // {
        //     Msg_warn("result_num and result_dim is: %zu, %zu...", Lvec.size(), K * query_num);
        //     for (int test_id = 0; test_id < Lvec.size(); test_id++)
        //     {
        //         for (int j = 0; j < K * query_num; ++j)
        //         {
        //             Msg_major("after recv before vector in pro 1: result_ids[%d][%d] = %d", test_id, j,
        //                       result_ids0[test_id][j]);
        //         }
        //     }
        // }

        // auto after_recv1 = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double> recv1_time = after_recv1 - after_recv2;
        // Msg_warn("----------------------recv 3 time is: %f", recv1_time.count());

        // assert(result_num0 == result_num1);
        // assert(result_num1 == result_num2);

        //= result merge
        std::vector<std::vector<uint32_t>> res_id(result_num0);
        std::vector<std::vector<float>> res_dist(result_num0);

        // int lastLvec = Lvec.size() - 1;
        merge_to_vector1<uint32_t, float, uint32_t, float>(res_id, res_dist, result_ids0, result_dists0, result_num0,
                                                           result_dim0);

        // for (int ids_index = 0; ids_index < K; ++ids_index)
        // {
        //     Msg_info("after merge result_ids in Lvec %d, is: %d, and its dist is: %f", lastLvec,
        //              res_id[lastLvec][ids_index], );
        // }
        merge_to_vector2<uint32_t, float, uint32_t, float>(res_id, res_dist, result_ids1, result_dists1, result_num0,
                                                           result_dim0);
        merge_to_vector3<uint32_t, float, uint32_t, float>(res_id, res_dist, result_ids2, result_dists2, result_num0,
                                                           result_dim0);

        sort_vector<uint32_t, float>(res_id, res_dist, K, query_num);

        //= begin Lvec.size() loop----------------------------------------------
        double best_recall = 0.0;
        for (uint32_t test_id = 0; test_id < Lvec.size(); test_id++)
        {
            if (test_id == Lvec.size() - 1)
                for (int ids_index = 0; ids_index < K; ++ids_index)
                    printf("after sort result_ids in Lvec %d, is: %d\n", test_id, res_id[test_id][ids_index]);
            std::cout << std::endl;
            double recall = 0;
            if (calc_recall_flag)
            {
                //* 计算recall函数，返回值为recall，对每个L都计算一次。
                //* 传入参数为：query_num 查询个数、gt_ids, gt_dists, gt_dim,
                //* query_result_ids[test_id]这个是一维数组首地址，recall_at 就是K，所以参数传递没有问题。
                recall = diskann::calculate_recall((uint32_t)query_num, gt_ids, gt_dists, (uint32_t)gt_dim,
                                                   res_id[test_id].data(), K, K);
                // 在L个结果中，选出最佳recall
                best_recall = std::max(recall, best_recall);
            }
            if (calc_recall_flag)
            {
                diskann::cout << std::setw(16) << recall << std::endl;
            }
            else
                diskann::cout << std::endl;
        } // end of Lvec.size()
        auto dis_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> total_dis_time = dis_time - s;
        std::cout << "----------------------total single time is: " << total_dis_time.count() << std::endl;
    } // end of serverId()=0

    //*======================================================================================================================*/
    //*
    //*                                                     server1
    //*
    //*======================================================================================================================*/

    else
    {
        // try
        // {
        uint32_t query_buffer_capacity = (1000 * 960 * sizeof(data_type) + 2 * sizeof(size_t)) * 2;
        uint32_t result_buffer_capacity = (K * Lvec.size() * 1000 * sizeof(data_type) + 2 * sizeof(size_t)) * 1.2;

        Message::MessageQueue<char> *msgq1 =
            new Message::MessageQueue<char>(result_buffer_capacity, query_buffer_capacity);
        size_t query_num, query_dim = 0;
        float *recv_query = nullptr;
        uint32_t **result_ids = nullptr;
        float **result_dists = nullptr;

        msgq1->recvMessage(0, recv_query, query_num, query_dim, QUERY);

        if (!query_filters.empty() && label_type == "ushort")
        {
            if (data_type == std::string("float"))
            {
                return search_disk_index<float, uint16_t>(
                    metric, index_path_prefix, result_path_prefix, query_file, gt_file, num_threads, K, W,
                    num_nodes_to_cache, search_io_limit, Lvec, fail_if_recall_below, query_filters, use_reorder_data);
            }

            else if (data_type == std::string("int8"))
                return search_disk_index<int8_t, uint16_t>(
                    metric, index_path_prefix, result_path_prefix, query_file, gt_file, num_threads, K, W,
                    num_nodes_to_cache, search_io_limit, Lvec, fail_if_recall_below, query_filters, use_reorder_data);
            else if (data_type == std::string("uint8"))
                return search_disk_index<uint8_t, uint16_t>(
                    metric, index_path_prefix, result_path_prefix, query_file, gt_file, num_threads, K, W,
                    num_nodes_to_cache, search_io_limit, Lvec, fail_if_recall_below, query_filters, use_reorder_data);
            else
            {
                std::cerr << "Unsupported data type. Use float or int8 or uint8" << std::endl;
                return -1;
            }
        }
        else
        {
            if (data_type == std::string("float"))
            {
                search_disk_index_mpi<float>(metric, index_path_prefix, result_path_prefix, result_ids, result_dists,
                                             recv_query, query_num, query_dim, gt_file, num_threads, K, W,
                                             num_nodes_to_cache, search_io_limit, Lvec, fail_if_recall_below,
                                             query_filters, use_reorder_data);
                // std::cout << "----------------------total single time is: " << diff.count() << std::endl;


                // return search_disk_index<float>(metric, index_path_prefix, result_path_prefix, query_file, gt_file,
                //                                 num_threads, K, W, num_nodes_to_cache, search_io_limit, Lvec,
                //                                 fail_if_recall_below, query_filters, use_reorder_data);
                msgq1->sendMessage<uint32_t>(0, result_ids, Lvec.size(), K * query_num, IDX);
                msgq1->sendMessage<float>(0, result_dists, Lvec.size(), K * query_num, DIST);
            }
            else if (data_type == std::string("int8"))
                search_disk_index<int8_t>(metric, index_path_prefix, result_path_prefix, query_file, gt_file,
                                          num_threads, K, W, num_nodes_to_cache, search_io_limit, Lvec,
                                          fail_if_recall_below, query_filters, use_reorder_data);
            else if (data_type == std::string("uint8"))
                search_disk_index<uint8_t>(metric, index_path_prefix, result_path_prefix, query_file, gt_file,
                                           num_threads, K, W, num_nodes_to_cache, search_io_limit, Lvec,
                                           fail_if_recall_below, query_filters, use_reorder_data);
            else
            {
                std::cerr << "Unsupported data type. Use float or int8 or uint8" << std::endl;
                Env::endEnv();
                return -1;
            }
        }
    } // end of serverId()=1
    Env::endEnv();
    // catch (const std::exception &e)
    // {
    //     std::cout << std::string(e.what()) << std::endl;
    //     diskann::cerr << "Index search failed." << std::endl;
    //     return -1;
    // }
}